#include "hip/hip_runtime.h"
//#include <TH/TH.h>
#include <stdbool.h>
#include <stdio.h>
#include "my_lib_invert_cuda_kernel.h"
#define real float

__device__  void dot43(real A[4][3], real B[3][3]) {
    int i,j,k;
    for (i = 0; i<3; i++)
    {
            for (j = 0; j<3; j++) {
                B[i][j] = 0;
                for (k = 0; k < 4; k++)
                    B[i][j] += A[k][i] * A[k][j];
        //printf("%f ", B[i][j]);
        }
        //printf("\n");
    }
    //printf("\n");
}


__device__  void inv3(real B[3][3], real invB[3][3]) {
    float determinant = +B[0][0]*(B[1][1]*B[2][2]-B[2][1]*B[1][2])
                        -B[0][1]*(B[1][0]*B[2][2]-B[1][2]*B[2][0])
                        +B[0][2]*(B[1][0]*B[2][1]-B[1][1]*B[2][0]);
    float invdet = 1/determinant;
    
    //printf("det %f\n", determinant);
    invB[0][0] =  (B[1][1]*B[2][2]-B[2][1]*B[1][2])*invdet;
    invB[1][0] = -(B[0][1]*B[2][2]-B[0][2]*B[2][1])*invdet;
    invB[2][0] =  (B[0][1]*B[1][2]-B[0][2]*B[1][1])*invdet;
    invB[0][1] = -(B[1][0]*B[2][2]-B[1][2]*B[2][0])*invdet;
    invB[1][1] =  (B[0][0]*B[2][2]-B[0][2]*B[2][0])*invdet;
    invB[2][1] = -(B[0][0]*B[1][2]-B[1][0]*B[0][2])*invdet;
    invB[0][2] =  (B[1][0]*B[2][1]-B[2][0]*B[1][1])*invdet;
    invB[1][2] = -(B[0][0]*B[2][1]-B[2][0]*B[0][1])*invdet;
    invB[2][2] =  (B[0][0]*B[1][1]-B[1][0]*B[0][1])*invdet;
        
}
    

__device__  void dot34(real invB[3][3], real A[4][3], real m[3][4]) {
    int i, j, k;
    for (i = 0; i < 3; i++)
        for (j = 0; j < 4; j++){
            m[i][j] = 0;
            for (k = 0; k < 3; k++) {
                m[i][j] += invB[i][k] * A[j][k];
            }
    }
}


__device__  void dot41(real m[3][4], real x[4], real alpha[3]) {
    int i,j;
    for (i = 0; i < 3; i++) {
        alpha[i] = 0;
        for (j = 0; j < 4; j++)
            alpha[i] += m[i][j] * x[j];
         //printf("%.2f ", alpha[i]);
    }
    //printf("\n");
}

__device__  real min(real * array, int len) {
    real m = array[0];
    int i;
    for (int i = 0; i < len; i++) 
        if (array[i] < m) m = array[i];
    return m;
}

__device__  real max(real * array, int len) {
    real m = array[0];
    int i;
    for (int i = 0; i < len; i++) 
        if (array[i] > m) m = array[i];
    return m;
}


__device__  void dot21(real im2[2][2], real d[2], real r[2]) {
    int i,j;
    for (i = 0; i < 2; i++) {
        r[i] = 0;
        for (j = 0; j < 2; j++)
            r[i] += im2[i][j] * d[j];
    }
}



__device__  void dot22(real m1[2][2], real m2[2][2], real result[2][2]) {
    int i,j,k;
    for (i = 0; i < 2; i++ )
        for (j = 0; j < 2; j++)
        {
            result[i][j] = 0;
            for (k = 0; k < 2; k++)
                result[i][j] += m1[i][k] * m2[k][j];
        }
}


__device__  void dot32(real gradalphar[3][2], real gradr[2], real gradalpha[3]) {
    int i,j;
    for (i = 0; i < 3; i++) {
        gradalpha[i] = 0;
        for (j = 0; j < 2; j++) 
            gradalpha[i] += gradalphar[i][j] * gradr[j];
        }
}


__device__  void inv2(real m2[2][2], real im2[2][2]) {
   real determinant = m2[0][0] * m2[1][1] - m2[0][1] * m2[1][0];
   //printf("det %.5f\n", determinant);
   im2[0][0] = m2[1][1] / determinant;
   im2[1][1] = m2[0][0] / determinant;
   im2[0][1] = -m2[0][1] / determinant;
   im2[1][0] = -m2[1][0] / determinant;
}

__device__  void dot34t(real m[3][4], real alpha[3], real gradx[4]) {
    int i,j;
    for (i = 0; i < 4; i++) {
        gradx[i] = 0;
        for (j = 0; j < 3; j++) 
            gradx[i] += m[j][i] * alpha[j];
    }
}

__device__  real abs_real(real num) {
    return (num > 0)?num:-num;
}
   
__global__ void test(float * a, float * b, int c, int d) {
	
} 
__global__ void InvSamplerBHWD_updateOutput(//(THFloatTensor *inputImages, THFloatTensor *grids, THFloatTensor *invgrids, THFloatTensor *output, THFloatTensor *depth_map)
  int batchsize 			,//= inputImages->size[0];
  int inputImages_height 	,//= inputImages->size[1];
  int inputImages_width	 	,//= inputImages->size[2];
  int output_height 		,//= output->size[1];
  int output_width 			,//= output->size[2];
  int inputImages_channels 	,//= inputImages->size[3];
  int output_strideBatch 	,//= output->stride[0];
  int output_strideHeight 	,//= output->stride[1];
  int output_strideWidth 	,//= output->stride[2];
  int depth_strideBatch 	,//= depth_map->stride[0];
  int depth_strideHeight 	,//= depth_map->stride[1];
  int depth_strideWidth 	,//= depth_map->stride[2];
  int inputImages_strideBatch 	,//= inputImages->stride[0];
  int inputImages_strideHeight 	,//= inputImages->stride[1];
  int inputImages_strideWidth 	,//= inputImages->stride[2];
  int grids_strideBatch 	,//= grids->stride[0];
  int grids_strideHeight 	,//= grids->stride[1];
  int grids_strideWidth 	,//= grids->stride[2];
  float *inputImages_data, float *output_data, float *grids_data, float *invgrids_data, float *depth_data,
  float *target_depth_data) //= (real *)malloc(sizeof(real) * output_height * output_width * batchsize);
  //inputImages_data = THFloatTensor_data(inputImages);
  //output_data = THFloatTensor_data(output);
  //grids_data = THFloatTensor_data(grids);
  //invgrids_data = THFloatTensor_data(invgrids);
  //depth_data = THFloatTensor_data(depth_map);  
  {
 
  int tradeb, yOut, xOut, k;

  real x[4], y[4], alpha[3], beta[3];
    
  real m2[2][2], im2[2][2];
  
  real minx, miny, minbasex, minbasey;
  real maxx, maxy, maxbasex, maxbasey;
    
  int b;
  
  for(b=0; b < batchsize; b++)
  {
    for(yOut=0; yOut < output_height - 1; yOut++)
    {
      for(xOut=0; xOut < output_width - 1; xOut++) {
          const int outdepthAddress = depth_strideBatch * b + depth_strideHeight * yOut + depth_strideWidth * xOut;
          target_depth_data[outdepthAddress] = 1e5;
      }
    }
  }
   
  for(b=0; b < batchsize; b++)
  {
    for(yOut=0; yOut < output_height - 1; yOut++)
    {
      for(xOut=0; xOut < output_width - 1; xOut++)
      {
        //read the grid
          

          const int inTopLeftAddress = grids_strideBatch * b + grids_strideHeight * yOut + grids_strideWidth * xOut;
          const int inTopRightAddress = inTopLeftAddress + grids_strideWidth;
          const int inBottomLeftAddress = inTopLeftAddress + grids_strideHeight;
          const int inBottomRightAddress = inBottomLeftAddress + grids_strideWidth;

          
        x[0] = grids_data[inTopLeftAddress + 1];
        x[1] = grids_data[inBottomLeftAddress + 1];
        x[2] = grids_data[inTopRightAddress + 1];
        x[3] = grids_data[inBottomRightAddress + 1];
          
        y[0] = grids_data[inTopLeftAddress];
        y[1] = grids_data[inBottomLeftAddress];
        y[2] = grids_data[inTopRightAddress];
        y[3] = grids_data[inBottomRightAddress];
          
          
        //if (abs_real(x[2] - x[0]) > 1) {
        //    if (x[0] < 0) x[0] += 2;
        //    if (x[2] < 0) x[2] += 2;
        //}
        //  
        //if (abs_real(x[3] - x[1]) > 1) {
        //    if (x[1] < 0) x[1] += 2;
        //    if (x[3] < 0) x[3] += 2;
        //}
        
          
        float dx1 = x[2] - x[0];
        float dy1 = y[2] - y[0];
         
        float dx2 = x[1] - x[0];
        float dy2 = y[1] - y[0];
        float normal = (dx1 * dy2) - (dx2 * dy1);
          
        
        real m[3][4] = {{ 0.7500,    0.2500,    0.2500,   -0.2500},{-0.5000,   -0.5000,    0.5000,    0.5000},{-0.5000,    0.5000,   -0.5000,    0.5000}};  
         
        dot41(m, x, alpha);
        dot41(m, y, beta);
              
        //printf("recon %.4f = %.4f\n", A[0][0] * alpha[0] + A[0][1] * alpha[1] + A[0][2] * alpha[2], x[0]);
        //printf("%.2f %.2f %.2f %.2f %.2f %.2f\n", alpha[0], alpha[1], alpha[2], beta[0], beta[1], beta[2]);    
             
        minx = min(x, 4);
        miny = min(y, 4);
        maxx = max(x, 4);
        maxy = max(y, 4);

         
        int minxcoord = floor((minx + 1) * (inputImages_width - 1)  / 2);
        int maxxcoord = ceil((maxx + 1) * (inputImages_width - 1)  / 2);
          
        int minycoord = floor((miny + 1) * (inputImages_height - 1)  / 2);
        int maxycoord = ceil((maxy + 1) * (inputImages_height - 1)  / 2);
         
        //printf("%d %d %d %d\n", minxcoord, maxxcoord, minycoord, maxycoord);
          
        m2[0][0] = alpha[1];
        m2[0][1] = alpha[2];
        m2[1][0] = beta[1];
        m2[1][1] = beta[2];
        
        inv2(m2, im2);
         
        //printf("%.2f, %.2f \n%.2f, %.2f \n\n", im2[0][0], im2[0][1], im2[1][0], im2[1][1]);
    
        int xcoord, ycoord; 
         
        int scaling = (maxxcoord - minxcoord) *  (maxycoord - minycoord);
          
        if (normal > 0)
        if ((maxxcoord - minxcoord) < inputImages_width / 2)
            for (xcoord = minxcoord;  xcoord < maxxcoord; xcoord ++)
                for (ycoord = minycoord; ycoord < maxycoord; ycoord ++) {
                     
                    real d2[2];
                    real yf = (float)ycoord / (float)(output_height-1) * 2 - 1;
                    real xf = (float)xcoord / (float)(output_width-1) * 2 - 1;
                    
                    d2[0] = xf - alpha[0];
                    d2[1] = yf - beta[0];
                    
                    real r[2];
                    dot21(im2, d2, r); // r[0] x, r[1] y;
            
                    real slack = 0;
                    //printf("%f %f\n", r[0], r[1]);
                    //printf("%.4f = %.4f\n", alpha[0] + alpha[1] * r[0] + alpha[2] * r[1], xf);
                    if ((-slack < r[0]) && (r[0] < 1+slack) &&(-slack < r[1]) && (r[1] < 1 + slack)) {
                        //printf("%.4f, %.4f | %.4f %.4f \n", r[0], r[1], basex[0], basey[0]);
                        int yInTopLeft, xInTopLeft;
                        real yWeightTopLeft, xWeightTopLeft;

                        real xcoord_source = r[0] + xOut;
                        xInTopLeft = floor(xcoord_source);
                        xWeightTopLeft = 1 - (xcoord_source - xInTopLeft);

                        real ycoord_source = r[1] + yOut;
                        yInTopLeft = floor(ycoord_source);
                        yWeightTopLeft = 1 - (ycoord_source - yInTopLeft);
                        
                        const int outAddress = output_strideBatch * b + output_strideHeight * ycoord + output_strideWidth * xcoord;
                        const int outGridAddress = grids_strideBatch * b + grids_strideHeight * ycoord + grids_strideWidth * xcoord;
                        const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;
                        const int inTopRightAddress = inTopLeftAddress + inputImages_strideWidth;
                        const int inBottomLeftAddress = inTopLeftAddress + inputImages_strideHeight;
                        const int inBottomRightAddress = inBottomLeftAddress + inputImages_strideWidth;
                        
                        const int indepthAddress = depth_strideBatch * b + depth_strideHeight * yInTopLeft + depth_strideWidth * xInTopLeft;
                        const int outdepthAddress = depth_strideBatch * b + depth_strideHeight * ycoord + depth_strideWidth * xcoord;
                                                                   
                        real v=0;
                        real inTopLeft=0;
                        real inTopRight=0;
                        real inBottomLeft=0;
                        real inBottomRight=0;
                        
                        bool topLeftIsIn = xInTopLeft >= 0 && xInTopLeft <= inputImages_width-1 && yInTopLeft >= 0 && yInTopLeft <= inputImages_height-1;
                        bool topRightIsIn = xInTopLeft+1 >= 0 && xInTopLeft+1 <= inputImages_width-1 && yInTopLeft >= 0 && yInTopLeft <= inputImages_height-1;
                        bool bottomLeftIsIn = xInTopLeft >= 0 && xInTopLeft <= inputImages_width-1 && yInTopLeft+1 >= 0 && yInTopLeft+1 <= inputImages_height-1;
                        bool bottomRightIsIn = xInTopLeft+1 >= 0 && xInTopLeft+1 <= inputImages_width-1 && yInTopLeft+1 >= 0 && yInTopLeft+1 <= inputImages_height-1;

                        bool outIsIn =  xcoord >= 0 && xcoord <= inputImages_width-1 && ycoord >= 0 && ycoord <= inputImages_height-1;
                        
                        int t;          
                        
                        for(t=0; t<inputImages_channels; t++)
                        {
                           if(topLeftIsIn) inTopLeft = inputImages_data[inTopLeftAddress + t];
                           if(topRightIsIn) inTopRight = inputImages_data[inTopRightAddress + t];
                           if(bottomLeftIsIn) inBottomLeft = inputImages_data[inBottomLeftAddress + t];
                           if(bottomRightIsIn) inBottomRight = inputImages_data[inBottomRightAddress + t];

                           v = xWeightTopLeft * yWeightTopLeft * inTopLeft
                             + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
                             + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
                             + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;

                           if (scaling < 36){
                           if (outIsIn)
                           if ((depth_data[indepthAddress] < target_depth_data[outdepthAddress]) && (depth_data[indepthAddress] > 0))
                               output_data[outAddress + t] = v;
                           }
                            else {
                                if (outIsIn) {
                                output_data[outAddress + t] = 0;
                                target_depth_data[outdepthAddress] = 0;
                                }
                            }
                        
                        }
                        
                        if (outIsIn)
                        if ((depth_data[indepthAddress] < target_depth_data[outdepthAddress]) && (depth_data[indepthAddress] > 0)) {
                             invgrids_data[outGridAddress] = (float)yOut;
                             invgrids_data[outGridAddress+1] = (float)xOut; // x - [+1], y - [0]
                        }
                        
                        if (outIsIn)
                        if ((depth_data[indepthAddress] < target_depth_data[outdepthAddress]) && (depth_data[indepthAddress] > 0)) {
                            target_depth_data[outdepthAddress] = depth_data[indepthAddress];
                        }
                    } 
        }

      }
    }
  }

  //free(target_depth_data);
  return;
}

__global__ void InvSamplerBHWD_updateGradInput//(THFloatTensor *inputImages, THFloatTensor *grids, THFloatTensor *invgrids, THFloatTensor *gradInputImages, THFloatTensor *gradGrids, THFloatTensor *gradOutput)
(
  int batchsize 					,//= inputImages->size[0];
  int inputImages_height 			,//= inputImages->size[1];
  int inputImages_width 			,//= inputImages->size[2];
  int gradOutput_height 			,//= gradOutput->size[1];
  int gradOutput_width 				,//= gradOutput->size[2];
  int inputImages_channels 			,//= inputImages->size[3];
  int gradOutput_strideBatch 		,//= gradOutput->stride[0];
  int gradOutput_strideHeight 		,//= gradOutput->stride[1];
  int gradOutput_strideWidth 		,//= gradOutput->stride[2];
  int inputImages_strideBatch 		,//= inputImages->stride[0];
  int inputImages_strideHeight 		,//= inputImages->stride[1];
  int inputImages_strideWidth 		,//= inputImages->stride[2];
  int gradInputImages_strideBatch 	,//= gradInputImages->stride[0];
  int gradInputImages_strideHeight 	,//= gradInputImages->stride[1];
  int gradInputImages_strideWidth 	,//= gradInputImages->stride[2];
  int grids_strideBatch 			,//= grids->stride[0];
  int grids_strideHeight 			,//= grids->stride[1];
  int grids_strideWidth 			,//= grids->stride[2];
  int gradGrids_strideBatch 		,//= gradGrids->stride[0];
  int gradGrids_strideHeight 		,//= gradGrids->stride[1];
  int gradGrids_strideWidth 		,//= gradGrids->stride[2];
  float *inputImages_data, float *gradOutput_data,  float *grids_data,  float *gradGrids_data, float *gradInputImages_data, float *invgrids_data)
{
  //inputImages_data = THFloatTensor_data(inputImages);
  //gradOutput_data = THFloatTensor_data(gradOutput);
  //grids_data = THFloatTensor_data(grids);
  //invgrids_data = THFloatTensor_data(invgrids);
  //gradGrids_data = THFloatTensor_data(gradGrids);
  //gradInputImages_data = THFloatTensor_data(gradInputImages);
  bool onlyGrid=false;
  int b, yOut, xOut;
   
  for(b=0; b < batchsize; b++)
  {
    for(yOut=0; yOut < gradOutput_height; yOut++)
    {
      for(xOut=0; xOut < gradOutput_width; xOut++)
      {
          const int Address = gradGrids_strideBatch * b + gradGrids_strideHeight * yOut + gradGrids_strideWidth * xOut;
          gradGrids_data[Address] = 0;
          gradGrids_data[Address + 1] = 0;
          
      }
    }
  }
    
   for(b=0; b < batchsize; b++)
  {
    for(yOut=0; yOut < gradOutput_height; yOut++)
    {
      for(xOut=0; xOut < gradOutput_width; xOut++)
      {
          const int gradOutputAddress = gradOutput_strideBatch * b + gradOutput_strideHeight * yOut + gradOutput_strideWidth * xOut;
          const int invgridAddress = grids_strideBatch * b + grids_strideHeight * yOut + grids_strideWidth * xOut;
          
          real r[2], gradr[2];
          
          int xSource, ySource;
          
          xSource = (int)invgrids_data[invgridAddress + 1];
          ySource = (int)invgrids_data[invgridAddress];
          
          //printf("%d %d\n", xSource ,ySource);
          
          const int gridinTopLeftAddress = grids_strideBatch * b + grids_strideHeight * ySource + grids_strideWidth * xSource;
          const int gridinTopRightAddress = gridinTopLeftAddress + grids_strideWidth;
          const int gridinBottomLeftAddress = gridinTopLeftAddress + grids_strideHeight;
          const int gridinBottomRightAddress = gridinBottomLeftAddress + grids_strideWidth;
          
          int i,j;
          
          real m[3][4] = {{ 0.7500,    0.2500,    0.2500,   -0.2500},{-0.5000,   -0.5000,    0.5000,    0.5000},{-0.5000,    0.5000,   -0.5000,    0.5000}};  
          
          real gradalpha[3], gradbeta[3], alpha[3], beta[3];
         
          real x[4], y[4];
          x[0] = grids_data[gridinTopLeftAddress + 1];
          x[1] = grids_data[gridinBottomLeftAddress + 1];
          x[2] = grids_data[gridinTopRightAddress + 1];
          x[3] = grids_data[gridinBottomRightAddress + 1];

          y[0] = grids_data[gridinTopLeftAddress];
          y[1] = grids_data[gridinBottomLeftAddress];
          y[2] = grids_data[gridinTopRightAddress];
          y[3] = grids_data[gridinBottomRightAddress];
          
          dot41(m, x, alpha);
          dot41(m, y, beta);
          real target_yf, target_xf;
          target_yf = (float)yOut / (float)(inputImages_height - 1) * 2 - 1;
          target_xf = (float)xOut / (float)(inputImages_width - 1) * 2 - 1;
          
          real m2[2][2], im2[2][2];
          m2[0][0] = alpha[1];
          m2[0][1] = alpha[2];
          m2[1][0] = beta[1];
          m2[1][1] = beta[2];
          inv2(m2, im2);
          
          real d2[2];
          d2[0] = target_xf - alpha[0];
          d2[1] = target_yf - beta[0];
          
          real r2[2];
          
          dot21(im2, d2, r2);
          
          
          if ((xSource != 0) || (ySource != 0)) {
               //printf("%d %d %.8f %.8f\n", xSource ,ySource, r2[0], r2[1]);
              
                // get the weights for interpolation
                int yInTopLeft, xInTopLeft;
                real yWeightTopLeft, xWeightTopLeft;
                real xgrad,ygrad;
                
                xInTopLeft = xSource;
                xWeightTopLeft = r[0];

                yInTopLeft = ySource;
                yWeightTopLeft = r[1];

                const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;
                const int inTopRightAddress = inTopLeftAddress + inputImages_strideWidth;
                const int inBottomLeftAddress = inTopLeftAddress + inputImages_strideHeight;
                const int inBottomRightAddress = inBottomLeftAddress + inputImages_strideWidth;

                const int gradInputImagesTopLeftAddress = gradInputImages_strideBatch * b + gradInputImages_strideHeight * yInTopLeft + gradInputImages_strideWidth * xInTopLeft;
                const int gradInputImagesTopRightAddress = gradInputImagesTopLeftAddress + gradInputImages_strideWidth;
                const int gradInputImagesBottomLeftAddress = gradInputImagesTopLeftAddress + gradInputImages_strideHeight;
                const int gradInputImagesBottomRightAddress = gradInputImagesBottomLeftAddress + gradInputImages_strideWidth;

                const int gradOutputAddress = gradOutput_strideBatch * b + gradOutput_strideHeight * yOut + gradOutput_strideWidth * xOut;

                real topLeftDotProduct = 0;
                real topRightDotProduct = 0;
                real bottomLeftDotProduct = 0;
                real bottomRightDotProduct = 0;

                real v=0;
                real inTopLeft=0;
                real inTopRight=0;
                real inBottomLeft=0;
                real inBottomRight=0;

                // we are careful with the boundaries
                bool topLeftIsIn = xInTopLeft >= 0 && xInTopLeft <= inputImages_width-1 && yInTopLeft >= 0 && yInTopLeft <= inputImages_height-1;
                bool topRightIsIn = xInTopLeft+1 >= 0 && xInTopLeft+1 <= inputImages_width-1 && yInTopLeft >= 0 && yInTopLeft <= inputImages_height-1;
                bool bottomLeftIsIn = xInTopLeft >= 0 && xInTopLeft <= inputImages_width-1 && yInTopLeft+1 >= 0 && yInTopLeft+1 <= inputImages_height-1;
                bool bottomRightIsIn = xInTopLeft+1 >= 0 && xInTopLeft+1 <= inputImages_width-1 && yInTopLeft+1 >= 0 && yInTopLeft+1 <= inputImages_height-1;

                int t;

                for(t=0; t<inputImages_channels; t++)
                {
                   real gradOutValue = gradOutput_data[gradOutputAddress + t];
                   if(topLeftIsIn)
                   {
                      real inTopLeft = inputImages_data[inTopLeftAddress + t];
                      topLeftDotProduct += inTopLeft * gradOutValue;
                      if(!onlyGrid) gradInputImages_data[gradInputImagesTopLeftAddress + t] += xWeightTopLeft * yWeightTopLeft * gradOutValue;
                   }

                   if(topRightIsIn)
                   {
                      real inTopRight = inputImages_data[inTopRightAddress + t];
                      topRightDotProduct += inTopRight * gradOutValue;
                      if(!onlyGrid) gradInputImages_data[gradInputImagesTopRightAddress + t] += (1 - xWeightTopLeft) * yWeightTopLeft * gradOutValue;
                   }

                   if(bottomLeftIsIn)
                   {
                      real inBottomLeft = inputImages_data[inBottomLeftAddress + t];
                      bottomLeftDotProduct += inBottomLeft * gradOutValue;
                      if(!onlyGrid) gradInputImages_data[gradInputImagesBottomLeftAddress + t] += xWeightTopLeft * (1 - yWeightTopLeft) * gradOutValue;
                   }

                   if(bottomRightIsIn)
                   {
                      real inBottomRight = inputImages_data[inBottomRightAddress + t];
                      bottomRightDotProduct += inBottomRight * gradOutValue;
                      if(!onlyGrid) gradInputImages_data[gradInputImagesBottomRightAddress + t] += (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * gradOutValue;
                   }
                }

                ygrad = - xWeightTopLeft * topLeftDotProduct + xWeightTopLeft * bottomLeftDotProduct - (1-xWeightTopLeft) * topRightDotProduct + (1-xWeightTopLeft) * bottomRightDotProduct;
                xgrad = - yWeightTopLeft * topLeftDotProduct + yWeightTopLeft * topRightDotProduct - (1-yWeightTopLeft) * bottomLeftDotProduct + (1-yWeightTopLeft) * bottomRightDotProduct;

                
                 //printf("%f %f\n", xgrad, ygrad);
              
              gradr[0] = xgrad;
              gradr[1] = ygrad;
              
              
              real grad_alpha_r[3][2], grad_beta_r[3][2], gradalpha[3], gradbeta[3];

              m2[0][0] = alpha[1];
              m2[0][1] = alpha[2];
              m2[1][0] = beta[1];
              m2[1][1] = beta[2];
              inv2(m2, im2);

              d2[0] = target_xf - alpha[0];
              d2[1] = target_yf - beta[0];


              real i00[2][2] = {{1,0},{0,0}};
              real temp[2][2];
              real temp2[2][2], tempgrad[2];

              dot22(im2, i00, temp);
              dot22(temp, im2, temp2);
              dot21(temp2, d2, tempgrad);
              grad_alpha_r[1][0] = -tempgrad[0];
              grad_alpha_r[1][1] = -tempgrad[1];


              real i01[2][2] = {{0,1},{0,0}};
              dot22(im2, i01, temp);
              dot22(temp, im2, temp2);
              dot21(temp2, d2, tempgrad);
              grad_alpha_r[2][0] = -tempgrad[0];
              grad_alpha_r[2][1] = -tempgrad[1];


              real i10[2][2] = {{0,0},{1,0}};
              dot22(im2, i10, temp);
              dot22(temp, im2, temp2);
              dot21(temp2, d2, tempgrad);
              grad_beta_r[1][0] = -tempgrad[0];
              grad_beta_r[1][1] = -tempgrad[1];


              real i11[2][2] = {{0,0},{0,1}};
              dot22(im2, i11, temp);
              dot22(temp, im2, temp2);
              dot21(temp2, d2, tempgrad);
              grad_beta_r[2][0] = -tempgrad[0];
              grad_beta_r[2][1] = -tempgrad[1];

              real j0[2] = {1,0};
              dot21(im2, j0, tempgrad);
              grad_alpha_r[0][0] = -tempgrad[0];
              grad_alpha_r[0][1] = -tempgrad[1];

              real j1[2] = {0,1};
              dot21(im2, j1, tempgrad);
              grad_beta_r[0][0] = -tempgrad[0];
              grad_beta_r[0][1] = -tempgrad[1];


              dot32(grad_beta_r, gradr, gradbeta);
              dot32(grad_alpha_r, gradr, gradalpha);
              
              //printf("%.3f %.3f %.3f\n", gradbeta[0], gradbeta[1], gradbeta[2]);
              
              real gradx[4], grady[4];
              dot34t(m, gradalpha, gradx);
              dot34t(m, gradbeta, grady);


              //printf("x %.3f %.3f %.3f %.3f\n", gradx[0], gradx[1], gradx[2], gradx[3]);
              //printf("y %.3f %.3f %.3f %.3f\n", grady[0], grady[1], grady[2], grady[3]);
              
              gradGrids_data[gridinTopLeftAddress] += grady[0];
              gradGrids_data[gridinTopLeftAddress + 1] += gradx[0];
              
              gradGrids_data[gridinBottomLeftAddress] += grady[1];
              gradGrids_data[gridinBottomLeftAddress + 1] += gradx[1];
              
              gradGrids_data[gridinTopRightAddress] += grady[2];
              gradGrids_data[gridinTopRightAddress + 1] += gradx[2];
              
              gradGrids_data[gridinBottomRightAddress] += grady[3];
              gradGrids_data[gridinBottomRightAddress + 1] += gradx[3];
                   
          }
      }
    }
   }

  return;
}

#ifdef __cplusplus
extern "C" {
#endif


int InvSamplerBHWD_updateOutput_cuda_kernel(
	int batchsize 			,//= inputImages->size[0];
  int inputImages_height 	,//= inputImages->size[1];
  int inputImages_width	 	,//= inputImages->size[2];
  int output_height 		,//= output->size[1];
  int output_width 			,//= output->size[2];
  int inputImages_channels 	,//= inputImages->size[3];
  int output_strideBatch 	,//= output->stride[0];
  int output_strideHeight 	,//= output->stride[1];
  int output_strideWidth 	,//= output->stride[2];
  int depth_strideBatch 	,//= depth_map->stride[0];
  int depth_strideHeight 	,//= depth_map->stride[1];
  int depth_strideWidth 	,//= depth_map->stride[2];
  int inputImages_strideBatch 	,//= inputImages->stride[0];
  int inputImages_strideHeight 	,//= inputImages->stride[1];
  int inputImages_strideWidth 	,//= inputImages->stride[2];
  int grids_strideBatch 	,//= grids->stride[0];
  int grids_strideHeight 	,//= grids->stride[1];
  int grids_strideWidth 	,//= grids->stride[2];
  float *inputImages_data, float *output_data, float *grids_data, float *invgrids_data, float *depth_data,
  float *target_depth_data,
  hipStream_t stream
)
{
	InvSamplerBHWD_updateOutput<<<1, 1, 0, stream>>> (
	  batchsize 			,//= inputImages->size[0];
	  inputImages_height 	,//= inputImages->size[1];
	  inputImages_width	 	,//= inputImages->size[2];
	  output_height 		,//= output->size[1];
	  output_width 			,//= output->size[2];
	  inputImages_channels 	,//= inputImages->size[3];
	  output_strideBatch 	,//= output->stride[0];
	  output_strideHeight 	,//= output->stride[1];
	  output_strideWidth 	,//= output->stride[2];
	  depth_strideBatch 	,//= depth_map->stride[0];
	  depth_strideHeight 	,//= depth_map->stride[1];
	  depth_strideWidth 	,//= depth_map->stride[2];
	  inputImages_strideBatch 	,//= inputImages->stride[0];
	  inputImages_strideHeight 	,//= inputImages->stride[1];
	  inputImages_strideWidth 	,//= inputImages->stride[2];
	  grids_strideBatch 	,//= grids->stride[0];
	  grids_strideHeight 	,//= grids->stride[1];
	  grids_strideWidth 	,//= grids->stride[2];
	  inputImages_data, 
	  output_data, 
	  grids_data, 
	  invgrids_data, 
	  depth_data,
	  target_depth_data
	);
	
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("error in BilinearSampler.updateGradInput: %s\n", hipGetErrorString(err));
      //THError("aborting");
      return 0;
    }
    return 1;
}

int InvSamplerBHWD_updateGradInput_cuda_kernel(
    int batchsize 					,//= inputImages->size[0];
    int inputImages_height 			,//= inputImages->size[1];
    int inputImages_width 			,//= inputImages->size[2];
    int gradOutput_height 			,//= gradOutput->size[1];
    int gradOutput_width 				,//= gradOutput->size[2];
    int inputImages_channels 			,//= inputImages->size[3];
    int gradOutput_strideBatch 		,//= gradOutput->stride[0];
    int gradOutput_strideHeight 		,//= gradOutput->stride[1];
    int gradOutput_strideWidth 		,//= gradOutput->stride[2];
    int inputImages_strideBatch 		,//= inputImages->stride[0];
    int inputImages_strideHeight 		,//= inputImages->stride[1];
    int inputImages_strideWidth 		,//= inputImages->stride[2];
    int gradInputImages_strideBatch 	,//= gradInputImages->stride[0];
    int gradInputImages_strideHeight 	,//= gradInputImages->stride[1];
    int gradInputImages_strideWidth 	,//= gradInputImages->stride[2];
    int grids_strideBatch 			,//= grids->stride[0];
    int grids_strideHeight 			,//= grids->stride[1];
    int grids_strideWidth 			,//= grids->stride[2];
    int gradGrids_strideBatch 		,//= gradGrids->stride[0];
    int gradGrids_strideHeight 		,//= gradGrids->stride[1];
    int gradGrids_strideWidth 		,//= gradGrids->stride[2];
    float *inputImages_data, 
	float *gradOutput_data,  
	float *grids_data,  
	float *gradGrids_data, 
	float *gradInputImages_data, 
	float *invgrids_data,
	hipStream_t stream
){
		
	printf("%d %d %d %d\n", batchsize, inputImages_height, inputImages_width, inputImages_channels);
		
	InvSamplerBHWD_updateGradInput<<<1, 1, 0, stream>>> (
	    batchsize 					,//= inputImages->size[0];
	    inputImages_height 			,//= inputImages->size[1];
	    inputImages_width 			,//= inputImages->size[2];
	    gradOutput_height 			,//= gradOutput->size[1];
	    gradOutput_width 				,//= gradOutput->size[2];
	    inputImages_channels 			,//= inputImages->size[3];
	    gradOutput_strideBatch 		,//= gradOutput->stride[0];
	    gradOutput_strideHeight 		,//= gradOutput->stride[1];
	    gradOutput_strideWidth 		,//= gradOutput->stride[2];
	    inputImages_strideBatch 		,//= inputImages->stride[0];
	    inputImages_strideHeight 		,//= inputImages->stride[1];
	    inputImages_strideWidth 		,//= inputImages->stride[2];
	    gradInputImages_strideBatch 	,//= gradInputImages->stride[0];
	    gradInputImages_strideHeight 	,//= gradInputImages->stride[1];
	    gradInputImages_strideWidth 	,//= gradInputImages->stride[2];
	    grids_strideBatch 			,//= grids->stride[0];
	    grids_strideHeight 			,//= grids->stride[1];
	    grids_strideWidth 			,//= grids->stride[2];
	    gradGrids_strideBatch 		,//= gradGrids->stride[0];
	    gradGrids_strideHeight 		,//= gradGrids->stride[1];
	    gradGrids_strideWidth 		,//= gradGrids->stride[2];
	    inputImages_data, 
		gradOutput_data,  
		grids_data,  
		gradGrids_data, 
		gradInputImages_data, 
		invgrids_data
		);
			
	    hipError_t err = hipGetLastError();
	    if (err != hipSuccess) {
	      printf("error in BilinearSampler.updateGradInput: %s\n", hipGetErrorString(err));
	      //THError("aborting");
	      return 0;
	    }
	    return 1;	
		
}




#ifdef __cplusplus
}
#endif
